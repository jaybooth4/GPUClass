
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <ctime>
#include <stdlib.h>
#include <sys/time.h>
 
// Thread block size
#define BLOCK_SIZE 16
#define TILE_SIZE 32 

#define ROW 1024
#define COL 1024

// GPU Functions
void MM_Basic(float *a, float *b, float *c, int row, int col, int k);
__global__ void MM_Basic_kernel( float *devA, float *devB, float *devC, int row, int col, int k);

void MM_Improved(float *a, float *b, float *c, int row, int col, int k);
__global__ void MM_Improved_kernel( float *devA, float *devB, float *devC, int row, int col, int k);

/*
 * Main function
 */
int main(int argC, char** argV) {
        
        //
        // Setup
        //////////////////

	// Time Variables
	float time;
	hipEvent_t start, stop;
	hipEventCreate (&start);
	hipEventCreate (&stop);
	
	// Matrices
	float *a, *b;
        float *c_cpu, *c_gpu_basic, *c_gpu_improved;
		
	//Setting matrix parameters.
	int row = ROW;
	int col = COL;
	int   k = COL;
	int sum = 0;
	
	// Process input arguments (if specified)
	switch (argC) {
		case 2: {
	                row = atoi(argV[1]);
        	        col = row;
                	  k = col;
			break;
		}
		case 3: {
			row = atoi(argV[1]);
			col = atoi(argV[2]);
			  k = col;
			break;
		}
		default: {
			//Nothing
		}
	}
	
	//Setting host memory space.
	a               = (float *) malloc(row*k*sizeof(float));
	b               = (float *) malloc(k*col*sizeof(float));
	c_cpu           = (float *) malloc(row*col*sizeof(float));
	c_gpu_basic     = (float *) malloc(row*col*sizeof(float));
	c_gpu_improved  = (float *) malloc(row*col*sizeof(float));
	
	//Initializing [A] and [B] with random values from 1 to 10, and C to 0
	printf ("Initializing Matricies, could take some time...\n");
	for(int i = 0 ; i < row ; i++ ){
		for(int j = 0 ; j < k ; j++ ){
			a[i*k+j] = rand()%10;
		}
	}
	for(int i = 0 ; i < k ; i++ ){
		for(int j = 0 ; j < col ; j++ ){
			b[i*col+j] = rand()%10;
		}
	}
	for(int i = 0 ; i < k ; i++ ){
		for(int j = 0 ; j < col ; j++ ){
			c_cpu           [i*col+j] = 0;
			c_gpu_basic     [i*col+j] = 0;
			c_gpu_improved  [i*col+j] = 0;
		}
	}

        //
        // CPU Calculation
        //////////////////
        
	printf("Running sequential job.\n");
	hipEventRecord(start,0);
	for(int i = 0 ; i < row ; i++ ){
		for(int j = 0 ; j < col ; j++ ){
			sum = 0;
			for(int w = 0 ; w < k ; w++ ){
				sum += a[i*k+w] * b[w*col+j];
			}
			c_cpu[i*col+j] = sum;
		}
	}
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("\tSequential Job Time: %.2f ms\n", time);

	//
        // Basic GPU Calculation
        ////////////////////////
        
	printf("Running Basic parallel job.\n");
	
	hipEventRecord(start,0);
	MM_Basic(a, b, c_gpu_basic, row, col, k);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&time, start, stop);
	printf("\tBasic Parallel Job Time: %.2f ms\n", time);

	// Compares matrices to make sure answer is correct, initializes c for next kernel.
	bool error = false;
	for(int i = 0 ; i < k ; i++ ){
		for(int j = 0 ; j < col ; j++ ){
			if (c_cpu[i*col+j] != c_gpu_basic[i*col+j]) {
				printf("\tError: Starting at [%d][%d]\n", i, j);
				error = true;
			}
			if (error) break;
		}
		if (error) break;
	}
	if (!error) printf("\tNo errors found.\n");
	
        //
        // Improved GPU Calculation
        ////////////////////////
        
	printf("Running Improved parallel job.\n");
	
	hipEventRecord(start,0);
	MM_Improved(a, b, c_gpu_improved, row, col, k);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&time, start, stop);
	printf("\tImproved Parallel Job Time: %.2f ms\n", time);

	// Compares matrices to make sure answer is correct, initializes c for next kernel.
	error = false;
	for(int i = 0 ; i < k ; i++ ){
		for(int j = 0 ; j < col ; j++ ){
			if (c_cpu[i*col+j] != c_gpu_improved[i*col+j]) {
				printf("\tError: Starting at [%d][%d]\n", i, j);
				error = true;
			}
			if (error) break;
		}
		if (error) break;
	}
	if (!error) printf("\tNo errors found.\n");

        free (a);
        free (b);
        free (c_cpu);
        free (c_gpu_basic);
        free (c_gpu_improved);
}

void MM_Basic(float *a, float *b, float *c, int row, int col, int k) {
	
	hipEvent_t kernelstart, kernelstop;
	float time;
	hipEventCreate (&kernelstart);
	hipEventCreate (&kernelstop);
	
	int sizeA = row*k*sizeof(float);
	int sizeB = k*col*sizeof(float);
	int sizeC = row*col*sizeof(float);
	float *devA, *devB, *devC;
	
	hipMalloc((void**)&devA, sizeA);
	hipMalloc((void**)&devB, sizeB);
	hipMalloc((void**)&devC, sizeC);
	
	hipMemcpy(devA, a, sizeA, hipMemcpyHostToDevice);
	hipMemcpy(devB, b, sizeB, hipMemcpyHostToDevice);
	
	dim3 dimBlock(16, 16, 1);
	dim3 dimGrid((COL+dimBlock.x-1)/dimBlock.x, (ROW+dimBlock.y-1)/dimBlock.y, 1);
	
	hipEventRecord(kernelstart,0);
	MM_Basic_kernel<<<dimGrid, dimBlock>>>(devA, devB, devC, row, col, k);
	hipEventRecord(kernelstop,0);
	hipEventSynchronize(kernelstop);

	hipEventElapsedTime(&time, kernelstart, kernelstop);
	printf("\tKernel Job Time: %.2f ms\n", time);
	
	hipMemcpy(c, devC, sizeC, hipMemcpyDeviceToHost);
	
	//Freeing device matrices.
	hipFree(devA); hipFree(devB); hipFree(devC);
}

__global__ void MM_Basic_kernel( float *devA, float *devB, float *devC, int row, int col, int k) {
	int txID = blockIdx.x * blockDim.x + threadIdx.x;
	int tyID = blockIdx.y * blockDim.y + threadIdx.y;
	
	if ((txID < col) && (tyID < row)) {
		float Pvalue = 0;
		for(int w = 0 ; w < k ; w++) {
			Pvalue += devA[tyID*k+w] * devB[w*k+txID];
		}
		devC[tyID*k+txID] = Pvalue;
	}
}

void MM_Improved(float *a, float *b, float *c, int row, int col, int k){

        // Write Code here
        hipEvent_t kernelstart, kernelstop;
        float time;
        hipEventCreate (&kernelstart);
        hipEventCreate (&kernelstop);

        int sizeA = row*k*sizeof(float);
        int sizeB = k*col*sizeof(float);
        int sizeC = row*col*sizeof(float);
        float *devA, *devB, *devC;

        hipMalloc((void**)&devA, sizeA);
        hipMalloc((void**)&devB, sizeB);
        hipMalloc((void**)&devC, sizeC);

        hipMemcpy(devA, a, sizeA, hipMemcpyHostToDevice);
        hipMemcpy(devB, b, sizeB, hipMemcpyHostToDevice);

        dim3 dimBlock(32, 32, 1);
        dim3 dimGrid((COL+dimBlock.x-1)/dimBlock.x, (ROW+dimBlock.y-1)/dimBlock.y, 1);

        hipEventRecord(kernelstart,0);
        MM_Basic_kernel<<<dimGrid, dimBlock>>>(devA, devB, devC, row, col, k);
        hipEventRecord(kernelstop,0);
        hipEventSynchronize(kernelstop);

        hipEventElapsedTime(&time, kernelstart, kernelstop);
        printf("\tKernel Job Time: %.2f ms\n", time);

        hipMemcpy(c, devC, sizeC, hipMemcpyDeviceToHost);

        //Freeing device matrices.
        hipFree(devA); hipFree(devB); hipFree(devC);

        
}
__global__ void MM_Improved_kernel( float *devA, float *devB, float *devC, int row, int col, int k){
        // Write Code here
	
	__shared__ int shareBlockA[TILE_SIZE][TILE_SIZE];
	__shared__ int shareBlockB[TILE_SIZE][TILE_SIZE];

	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int Row = by * TILE_SIZE + ty;
	int Col = bx * TILE_SIZE + tx;
	
	float Cvalue = 0;
        for (int m = 0; m < row/TILE_SIZE; ++m) {
          shareBlockA[ty][tx] = devA[Row * row + (m*TILE_SIZE + tx)];
	      shareBlockB[ty][tx] = devB[Col + (m * TILE_SIZE + ty) * row];
	      __syncthreads();
	      for (int k = 0; k < TILE_SIZE; ++k)
 		    Cvalue += shareBlockA[ty][k] * shareBlockB[k][tx];
	      __syncthreads();
      }
      devC[Row*row+Col] = Cvalue;
} 
